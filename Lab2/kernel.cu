/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

	__shared__ float ds_A [TILE_SIZE][TILE_SIZE];
	__shared__ float ds_B [TILE_SIZE][TILE_SIZE];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * blockDim.y + ty;
	int Col = bx * blockDim.x + tx;

	float Pvalue = 0;


	for(int p = 0; p < (TILE_SIZE + k - 1)/TILE_SIZE; p++)
	{
		if(p*TILE_SIZE + tx < k && Row < m)
			ds_A[ty][tx] = A[Row*k + p*TILE_SIZE + tx];
		else
			ds_A[ty][tx] = 0.0;

		if(p*TILE_SIZE + ty < k && Col < n)
			ds_B[ty][tx] = B[(p*TILE_SIZE + ty)*n + Col];
		else
			ds_B[ty][tx] = 0.0;

		__syncthreads();
		
		for(int i = 0; i < TILE_SIZE; i++) Pvalue += ds_A[ty][i] * ds_B[i][tx];

		__syncthreads();

	}
	
	if(Row < m && Col < n)
		C[Row*n+Col] = Pvalue;
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	dim3 DimGrid((n + BLOCK_SIZE-1)/BLOCK_SIZE + 1, (m + BLOCK_SIZE + 1)/BLOCK_SIZE + 1, 1);
	dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
	mysgemm<<<DimGrid, DimBlock>>>(m,n,k, A,B,C);



}


